#include "hip/hip_runtime.h"
/*
Problem Spec: 
- Create a program that utilizes the NPP library. 
- Executes kernel using NPP library to execute simple operations
- Test harness executes two separate runs of each kernel

Theodore Dyer
Introduction to GPU Programming Spring 2022 (EN605.617.81)
Chance Pascale
4/13/2022
*/

// Note - following the provided boxfilter example file as reference. 

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#  pragma warning(disable:4819)
#endif

#include <ImagesCPU.h>
#include <ImagesNPP.h>
#include <ImageIO.h>
#include <Exceptions.h>

#include <string.h>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <npp.h>

#include <helper_string.h>
#include <hip/hip_runtime_api.h>

inline int cudaDeviceInit(int argc, const char **argv)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        std::cerr << "CUDA error: no devices supporting CUDA." << std::endl;
        exit(EXIT_FAILURE);
    }

    int dev = findCudaDevice(argc, argv);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    std::cerr << "hipSetDevice GPU" << dev << " = " << deviceProp.name << std::endl;

    checkCudaErrors(hipSetDevice(dev));

    return dev;
}

int main(int argc, char *argv[])
{
    printf("%s Starting boxfilter conversion...\n\n", argv[0]);

    try
    {
        std::string sFilename;
        char *filePath;

        cudaDeviceInit(argc, (const char **)argv);

        if (checkCmdLineFlag(argc, (const char **)argv, "input"))
        {
            getCmdLineArgumentString(argc, (const char **)argv, "input", &filePath);
        }
        else
        {
            filePath = sdkFindFilePath("ghost.pgm", argv[0]);
        }

        if (filePath)
        {
            sFilename = filePath;
        }
        else
        {
            sFilename = "ghost.pgm";
        }

        // if we specify the filename at the command line, then we only test sFilename[0].
        int file_errors = 0;
        std::ifstream infile(sFilename.data(), std::ifstream::in);

        if (infile.good())
        {
            std::cout << "boxFilterNPP opened: <" << sFilename.data() << "> successfully!" << std::endl;
            file_errors = 0;
            infile.close();
        }
        else
        {
            std::cout << "boxFilterNPP unable to open: <" << sFilename.data() << ">" << std::endl;
            file_errors++;
            infile.close();
        }

        if (file_errors > 0)
        {
            exit(EXIT_FAILURE);
        }

        std::string sResultFilename = sFilename;

        std::string::size_type dot = sResultFilename.rfind('.');

        if (dot != std::string::npos)
        {
            sResultFilename = sResultFilename.substr(0, dot);
        }

        sResultFilename += "_boxFilter.pgm";

        if (checkCmdLineFlag(argc, (const char **)argv, "output"))
        {
            char *outputFilePath;
            getCmdLineArgumentString(argc, (const char **)argv, "output", &outputFilePath);
            sResultFilename = outputFilePath;
        }

        // declare a host image object for an 8-bit grayscale image
        npp::ImageCPU_8u_C1 oHostSrc;
        // load gray-scale image from disk
        npp::loadImage(sFilename, oHostSrc);
        // declare a device image and copy construct from the host image,
        // i.e. upload host to device
        npp::ImageNPP_8u_C1 oDeviceSrc(oHostSrc);

        // create struct with box-filter mask size
        NppiSize oMaskSize = {5, 5};

        NppiSize oSrcSize = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
        NppiPoint oSrcOffset = {0, 0};

        // create struct with ROI size
        NppiSize oSizeROI = {(int)oDeviceSrc.width() , (int)oDeviceSrc.height() };
        // allocate device image of appropriately reduced size
        npp::ImageNPP_8u_C1 oDeviceDst(oSizeROI.width, oSizeROI.height);
        // set anchor point inside the mask to (oMaskSize.width / 2, oMaskSize.height / 2)
        // It should round down when odd
        NppiPoint oAnchor = {oMaskSize.width / 2, oMaskSize.height / 2};

        // run box filter
        NPP_CHECK_NPP (
                           nppiFilterBoxBorder_8u_C1R(oDeviceSrc.data(), oDeviceSrc.pitch(),
                                                      oSrcSize, oSrcOffset,
                                                      oDeviceDst.data(), oDeviceDst.pitch(),
                                                      oSizeROI, oMaskSize, oAnchor, NPP_BORDER_REPLICATE) );

        // declare a host image for the result
        npp::ImageCPU_8u_C1 oHostDst(oDeviceDst.size());
        // and copy the device result data into it
        oDeviceDst.copyTo(oHostDst.data(), oHostDst.pitch());

        saveImage(sResultFilename, oHostDst);
        std::cout << "Saved image: " << sResultFilename << std::endl;

        exit(EXIT_SUCCESS);
    }
    catch (npp::Exception &rException)
    {
        std::cerr << "Program error! The following exception occurred: \n";
        std::cerr << rException << std::endl;
        std::cerr << "Aborting." << std::endl;

        exit(EXIT_FAILURE);
    }
    catch (...)
    {
        std::cerr << "Program error! An unknow type of exception occurred. \n";
        std::cerr << "Aborting." << std::endl;

        exit(EXIT_FAILURE);
        return -1;
    }

    return 0;
}
