#include "hip/hip_runtime.h"
/*
Problem Spec: 
- Executes kernel using CUDA streams and events to execute 4 math operations on data that 
    is fed into the kernel from the host code while it is running
- Test harness executes two separate runs of each kernel using CUDA streams and events

Theodore Dyer
Introduction to GPU Programming Spring 2022 (EN605.617.81)
Chance Pascale
3/14/2020

*/

#include <stdio.h> 
#include <time.h>
#include <hip/hip_runtime.h> 

//#define sizeOfArray 1024*1024
#define sizeOfArray 64

__global__ void arr_add(int *da, int *db, int *dr) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < sizeOfArray) {
        dr[tid] = da[tid] + db[tid]
    }
}

int main(int argc, char **argv) {
    codaDeviceProp prop;
    int *host_a, *host_b, *host_r;
    int *device_a, *device_b, *device_r;
    int whichDevice;

    hipGetDeviceCount(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);

    hipEvent_t start, stop;
    float elapsed_time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Device Allocate
    hipMalloc((void**)& device_a, sizeOfArray * sizeof(*device_a));
    hipMalloc((void**)& device_b, sizeOfArray * sizeof(*device_b));
    hipMalloc((void**)& device_r, sizeOfArray * sizeof(*device_r));

    // Host Allocate
    hipHostAlloc((void **)&host_a, sizeOfArray*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_b, sizeOfArray*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **)&host_r, sizeOfArray*sizeof(int), hipHostMallocDefault);

    // Populate Data
    for(int i = 0; i < sizeOfArray; i++) {
        host_a[i] = rand()%10;
        host_b[i] = rand()%10;
    }

    hipEventRecord(start);

    // Set up data copy to device
    hipMemcpyAsync(device_a, host_a, sizeOfArray * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(device_b, host_b, sizeOfArray * sizeof(int), hipMemcpyHostToDevice, stream);

    // Kernel
    arr_add<<<sizeOfArray, 1, 1, stream>>>(device_a, device_b, device_r);

    hipMemcpyAsync(host_r, device_r, sizeOfArray * sizeof(int), hipMemcpyDeviceToHost, stream);

    for(int i = 0; i < sizeOfArray; i++) {
        printf("hosta = %d, hostb = %d, hostr = %d\n", host_a[i], host_b[i], host_r[i]);
    }

    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("\nResults...\n");
    printf("\n Size of array : %d \n", sizeOfArray);
    printf("\n Time taken: %3.1f ms \n", elapsedTime);

    // Free Allocated Memory
    hipHostFree(host_a);
    cudaFreehost(host_b);
    hipHostFree(host_result);
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_r);

    return 0;

}

/*
function()
- generates data to be later used in testing utilization of register variables.

- Params:
    host_data_ptr = pointer to array in host memory that will be filled with 
        values through execution of this function

- Return:
    Void, however upon return the array pointer to by
    'host_data_ptr' will be populated with values
*/